#include "hip/hip_runtime.h"
#include "EngineEditor/LightmapsTool/stdafx.h"

// To makro musi by� zdefiniowane przed zaincludowaniem DirectXMath (w LightmapWorkerCUDA.h).
// Biblioteka u�ywa domy�lnie zestawu instrukcji SSE2 i trzeba to wy��czy�.
#define _XM_NO_INTRINSICS_

#include "EngineEditor/LightmapsTool/LightmapWorkers/LightmapWorkerCUDA.h"
#include "EngineCore/ModelsManager/DefaultAssets.h"

#include <cstring>
#include <limits>
// Trzeba oddefiniowa�, je�eli chce si� korzysta� z numeric_limits
#undef max

#include "glm/gtc/matrix_transform.hpp"
#include "glm/gtc/type_ptr.hpp"

#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

//#include "Common/memory_leaks.h"

using namespace DirectX;

LightmapWorkerCUDA::LightmapWorkerCUDA( SceneData* sceneData )
	: LightmapWorker( sceneData )
{
	m_threshold = 0.04f;
}

// ============================================================================= //
// Funkcje pomocnicze - og�lne
// ============================================================================= //


/**@brief Pobiera dane o wierzcho�kach zgromadzone na karcie graficznej.

@todo Funkcja zostanie zaimplementowana w przysz�o�ci.

Dane o wierzcho�kach meshy s� zgromadzone ju� na karcie graficznej.
Ta funkcja ma umo�liwi� pobranie wska�nik�w na te dane z obiektu BufferObject, kt�ry
S�u�y opakowaniu obiekt�w DirectXowych (lub w przysz�o�ci tak�e innych API graficznych).

Na razie dla sp�jno�ci kodu jest zwracany obiekt thrust::device_vector<VertexNormalTexCord1>.
W przysz�o�ci b�dzie to prawdopodobnie go�y wska�nik na pami�� i reszt� aplikacje b�dzie trzeba dostosowa�.

Do pobierania wska�nik�w pos�u�� funkcje:
hipGraphicsMapResources
hipGraphicsResourceGetMappedPointer*/
thrust::device_vector<VertexNormalTexCord1> GetVerticiesFromGraphicAPI( BufferObject* buffer )
{
	return thrust::device_vector<VertexNormalTexCord1>();
}

/**@brief Zmapowane wierzcho�ki funkcj� GetVerticiesFromGraphicAPI trzeba "odmapowa�" po zako�czeniu przetwarzania.*/
void UnmapVerticiesFromGraphicAPI()
{

}


/**@brief Szybsze (miejmy nadziej�) mno�enie przez 3.*/
inline unsigned int mul3( unsigned int number )
{
	return ( number << 1 ) + number;
}

inline float fmax( float a, float b, float c )
{
	return fmax( a, fmax( b, c ) );
}

/**@brief Maksymalna warto�� trzech liczb.*/
int max3( float x1, float x2, float x3 )
{
	if( x1 > x2 )
		return x1 > x3 ? (int)x1 : (int)x3;
	else
		return x2 > x3 ? (int)x2 : (int)x3;
}

/**@brief Minimalna warto�� trzech liczb.*/
int min3( float x1, float x2, float x3 )
{
	if( x1 < x2 )
		return x1 < x3 ? (int)x1 : (int)x3;
	else
		return x2 < x3 ? (int)x2 : (int)x3;
}


// ============================================================================= //
// Funkcje pomocnicze - graficzne
// ============================================================================= //

/**@brief Funkcja znajduje trojk�t o najwi�kszej zgromadznej energii do wyemitowania.

Prawdopodobnie b�dzie to robione jak�� form� redukcji.*/
__host__ std::tuple<unsigned int, unsigned int, float> FindMaxEmission( thrust::device_vector<glm::vec3>& emissionLight, unsigned emissionLightSize )
{
	//typedef std::tuple<unsigned int, unsigned int, glm::vec3> Emission;

	//class CompareEmissionFunction : thrust::binary_function < Emission, Emission, Emission >
	//{
	//public:
	//	Emission operator()( Emission &left, Emission &right )
	//	{
	//		glm::vec3 leftEmission = std::get<2>( left );
	//		glm::vec3 rightEmission = std::get<2>( right );

	//		float maxL = fmax( leftEmission.x, leftEmission.y, leftEmission.z );
	//		float maxR = fmax( rightEmission.x, rightEmission.y, rightEmission.z );

	//		return ( maxL > maxR ) ? left : right;
	//	}
	//};

	//Emission initialValue = std::make_tuple( 0, 0, glm::vec3( -1.0f, -1.0f, -1.0f ) );
	//Emission foundValue = thrust::reduce( emissionLight, emissionLight + emissionLightSize, initialValue, CompareEmissionFunction() );

	//glm::vec3 emissionValue = std::get<2>( foundValue );
	//return std::make_tuple( std::get<0>( foundValue ), std::get<1>( foundValue ), fmax( emissionValue.x, emissionValue.y, emissionValue.z ) );
}

/**@brief Wylicza wsp�rz�dn� barycentryczn� punktu wzgl�dem kraw�dzie.
Wsp�rz�dne nie s� wyskalowane do jedynki. Nale�y je wyskalowa� kiedy si� wyliczy wszystkie.*/
__device__ static float BarycentricCoords( glm::vec2& vertex1, glm::vec2& vertex2, glm::ivec2& point )
{
	return ( vertex2.x - vertex1.x ) * ( point.y - vertex1.y ) - ( vertex2.y - vertex1.y ) * ( point.x - vertex1.x );
}

/*
@param[inout] receiver Tr�jk�t do zrzutowania.W tej zmiennej zwracany jest tr�jk�t wynikowy.
@return Zwraca wektor z g��boko�ciami dla ka�dego tr�jk�ta.Je�eli jaki� wierzcho�ek
znajduje si� za p�aszczyzn� rzutowania, funkcja zwraca(-1.0, -1.0, -1.0).W normalnej sytuacji
wszystkie wsp�rz�dne s� dodatnie. */
__device__ static glm::vec3 HemisphereCast( Triangle4& emiter, Triangle4& receiver, glm::mat4& emiterViewMatrix )
{
	// Transformujemy wierzcho�ki do uk�adu emitera. Dzi�ki temu xy jest pozycj� na p�aszczy�nie (na razie jeszcze nie jednostkowej)
	// a z jest odleg�o�ci� od emitera.
	// @todo Zastanowi� sie czy nie da si� tu mno�y� przez macierz 3x3.
	glm::vec4 centerToVertex1 = emiterViewMatrix * glm::vec4( receiver.vertex1, 1.0f );		// Poniewa� to s� wierzcho�ki, to ustawiamy komponent w na 1.
	glm::vec4 centerToVertex2 = emiterViewMatrix * glm::vec4( receiver.vertex2, 1.0f );
	glm::vec4 centerToVertex3 = emiterViewMatrix * glm::vec4( receiver.vertex3, 1.0f );

	// Wyliczenie d�ugo�ci od �rodka do wierzcho�k�w.
	float depth1 = glm::length( glm::vec3( centerToVertex1 ) );
	float depth2 = glm::length( glm::vec3( centerToVertex2 ) );
	float depth3 = glm::length( glm::vec3( centerToVertex3 ) );

	// Normalizacja wektor�w
	receiver.vertex1 = glm::vec3( centerToVertex1 / depth1 );
	receiver.vertex2 = glm::vec3( centerToVertex2 / depth2 );
	receiver.vertex3 = glm::vec3( centerToVertex3 / depth3 );

	// Po tym przekszta�ceniu wektor jest wyra�ony we wsp�rz�dnych p�aszczyzny. Sk�adowa z oznacza
	// odleg�o�� obiektu od p�aszczyzny. Uwaga! Nie jest to odleg�o�� od �rodka p�aszczyzny, ale
	// odleg�o�� rzutu prostopad�ego na t� p�aszczyzn�. Mimo wszystko taka g��boko�� pozwala na poprawne
	// wyznaczenie przys�aniania.
	glm::vec3 depthVector;
	depthVector.x = -centerToVertex1.z;
	depthVector.y = -centerToVertex2.z;
	depthVector.z = -centerToVertex3.z;

	return depthVector;
}

/**@brief Dodaje pod podanymi indeksami w chunku element addValue.

Do zaimplementowania*/
__device__ inline glm::vec3& LoadAddStore( glm::vec3* chunk, unsigned int i, unsigned int j, glm::vec4 addValue )
{
	//auto& lightValueRef = chunk[ i ].Get<glm::vec3>( j );
	//glm::vec4 lightValue = XMLoadFloat3( &lightValueRef );
	//lightValue = XMVectorAdd( addValue, lightValue );
	//XMStoreFloat3( &lightValueRef, lightValue );

	//return lightValueRef;
}

/**@brief Rasteryzuje podany tr�jk�t. Sprawdza czy jest bli�ej emitera od ostatnio
wpisanego tr�jk�ta na podstawie bufora g��boko�ci i je�eli tak, to zapisuje nowy tr�jk�t.

Rasteryzacja zrobiona mniej wi�cej na podstawie strony:
https://fgiesen.wordpress.com/2013/02/06/the-barycentric-conspirac/
Warto przeczyta� te� kilka temat�w pokrewnych. Na razie implementuj� wersj� niewydajn�.
Mo�e potem przerobi� na lepsz�. W ka�dym razie warto zauwa�y�, �e ten algorytm jest bardzo dobry
do zr�wnoleglania, mo�e da si� to jako� sprytnie wykorzysta�.

@param[in] triangle Tr�jk�t do rasteryzacji.
@param[in] depths Odleg�o�ci (g��boko�ci) kolejnych wierzcho�k�w tr�jk�ta od emitera. (Tu jest ma�e oszustwo,
zobaczymy jak b�dzie si� sprawdza�, odleg�o�ci s� liczone do �rodka emitera).
@param[in] chunkIdx Indeks MemoryChunka do wpisania do bufora indeks�w.
@param[in] triangleIdx Indeks tr�jk�ta, kt�ry trafi do bufora indeks�w.
@param[in] depthBuffer Bufor g��boko�ci.
@param[in] indexBuffer Bufor indeks�w.*/
__device__ static void RasterizeTriangle( const Triangle4& triangle,
										  glm::vec3* depths,
										  unsigned int chunkIdx,
										  unsigned int triangleIdx,
										  float* depthBuffer,
										  BufferIndexing* indexBuffer,
										  unsigned int m_depthResolution )
{
	glm::vec2 triangles[ 3 ];

	triangles[ 0 ].x = triangle.vertex1.x;
	triangles[ 0 ].y = triangle.vertex1.y;

	triangles[ 1 ].x = triangle.vertex2.x;
	triangles[ 1 ].y = triangle.vertex2.y;

	triangles[ 2 ].x = triangle.vertex3.x;
	triangles[ 2 ].y = triangle.vertex3.y;

	// Wyliczamy prostok�t ograniczaj�cy.
	int minX = min3( triangles[ 0 ].x, triangles[ 1 ].x, triangles[ 2 ].x );
	int minY = min3( triangles[ 0 ].y, triangles[ 1 ].y, triangles[ 2 ].y );
	int maxX = max3( triangles[ 0 ].x, triangles[ 1 ].x, triangles[ 2 ].x );
	int maxY = max3( triangles[ 0 ].y, triangles[ 1 ].y, triangles[ 2 ].y );

	// Iterujemy po wszystkich elementach prostok�ta ograniczaj�cego.
	// Sprawdzamy czy dany punkt nale�y do tr�jk�ta. Je�eli nale�y
	// zagl�damy do z bufora.
	glm::ivec2 point;
	for( point.y = minY; point.y <= maxY; point.y++ )
	{
		for( point.x = minX; point.x <= maxX; point.x++ )
		{
			float w0 = BarycentricCoords( triangles[ 1 ], triangles[ 2 ], point );
			float w1 = BarycentricCoords( triangles[ 2 ], triangles[ 0 ], point );
			float w2 = BarycentricCoords( triangles[ 0 ], triangles[ 1 ], point );

			if( w0 >= 0 && w1 >= 0 && w2 >= 0 )	// Warunek na to, �e punkt jest wewn�trz tr�jk�ta.
			{
				float sum = w0 + w1 + w2;
				w0 = w0 / sum;
				w1 = w1 / sum;
				w2 = w2 / sum;

				int index = point.y * m_depthResolution + point.x;
				//float pointInverseDepth = w0 / ((float*)depths)[ 0 ] + w1 / ((float*)depths)[ 1 ] + w2 / ((float*)depths)[ 2 ];
				//float pointDepth = 1.0f / pointInverseDepth;
				float pointDepth = ( (float*)depths )[ 0 ] * w0 + ( (float*)depths )[ 1 ] * w1 + ( (float*)depths )[ 2 ] * w2;
				float& bufferDepth = depthBuffer[ index ];
				if( bufferDepth > pointDepth && pointDepth > 0.0f )
				{
					BufferIndexing& indicies = indexBuffer[ index ];
					indicies.first = chunkIdx;
					indicies.second = triangleIdx;
					bufferDepth = pointDepth;
				}
			}
		}
	}

}

/**@brief Transformuje wierzcho�ki w przedziale [-1,1] do wsp�rzednych bufora g��boko�ci.*/
__device__ static void HemisphereViewport( Triangle4& receiver, unsigned int m_depthResolution )
{
	glm::vec3 depthResolution = glm::vec3( (float)m_depthResolution / 2.0f );
	
	receiver.vertex1 = receiver.vertex1 * depthResolution + depthResolution;
	receiver.vertex2 = receiver.vertex2 * depthResolution + depthResolution;
	receiver.vertex3 = receiver.vertex3 * depthResolution + depthResolution;
}

/**@brief Tworzy macierz przekszta�caj�c� wierzcho�ki do uk�adu odniesienia emitera.*/
__host__ glm::mat4 LightmapWorkerCUDA::EmiterViewMatrix( Triangle4& emiter )
{
	// �rodek ci�ko�ci tr�jk�ta przyjujemy za �rodek emitera.
	glm::vec3 emiterCenter = emiter.vertex1 + emiter.vertex2;
	emiterCenter = emiterCenter + emiter.vertex3;
	emiterCenter = emiterCenter + glm::vec3( 3.0f );

	// Wektor prostopad�y do p�aszczyzny, na kt�rej le�� tr�jk�ty.
	glm::vec3 edge12 = emiter.vertex2 - emiter.vertex1;
	glm::vec3 edge13 = emiter.vertex3 - emiter.vertex1;
	glm::vec3 normal = glm::normalize( glm::cross( edge12, edge13 ) );

	// Ustalamy dowolnie (byle ortogonalnie) kierunek "w g�r�".
	glm::vec3 upDirection = glm::normalize( glm::cross( edge12, normal ) );

	// Tworzymy macierz widoku.
	glm::mat4 viewMatrix = glm::lookAt( emiterCenter, normal, upDirection );

	return viewMatrix;
}

// ============================================================================= //
// Generation - kernele
// ============================================================================= //

// @todo Zast�pi� lepszym rozwi�zaniem ni� zmienna globalna.
unsigned int depthBufferSize = 0;

/**@brief Czy�ci bufory g��boko�ci i indeks�w.*/
__global__ static void KernelClearDepthIndex( float *depthBuffer, BufferIndexing *indexBuffer )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	depthBuffer[ i ] = std::numeric_limits<float>::max();
	indexBuffer[ i ].first = INVALID_INDEX;
	indexBuffer[ i ].second = INVALID_INDEX;
}

__global__ static void KernelDepthPass( VertexFormat* verticies, Size vertexMax, uint16 i,
										 float* depthBuffer, BufferIndexing* indexBuffer, Triangle4 emiterPosition, glm::mat4 emiterViewMatrix )
{
	// Pierwsza wsp�rz�dna wierzcho�ka
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if( j > vertexMax ) return;


	// Wyliczamy pozycj� odbiorcy w uk�adzie wsp�rz�dnych bufora g��boko�ci.
	Triangle4 receiverPosition( verticies + j );
	glm::vec3 receiverDepths = HemisphereCast( emiterPosition, receiverPosition, emiterViewMatrix );

	if( receiverDepths.x < 0.0f || receiverDepths.y < 0.0f || receiverDepths.z < 0.0f )
		return;  // Tr�jk�t jest cz�ciowo za p�aszczyzn�. Musimy go odrzuci�.

	HemisphereViewport( receiverPosition, depthBufferSize );
	RasterizeTriangle( receiverPosition, &receiverDepths, i, j, depthBuffer, indexBuffer, depthBufferSize );
}


#define MAX_MATERIALS 300
__constant__ glm::vec3 materials[ MAX_MATERIALS ];

__global__ static void KernelTransferPass( BufferIndexing *indexBuffer,
											glm::vec3* reachedLight, glm::vec3* emissionLight, glm::vec4 emitedLight )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	BufferIndexing indicies = indexBuffer[ i ];
	// Tylko piksele na kole powinny dosta� dawk� energii, a bufor jest kwadratem.
	// Dlatego elementy, na kt�re nie zosta� zrzutowany �aden tr�jk�t zosta�y zainicjowane warto�ci� INVALID_INDEX.
	if( indicies.first != INVALID_INDEX )
	{
		glm::vec3 materialDiffuse = materials[ indicies.first ];
		glm::vec3 receivedLight = materialDiffuse * glm::vec3( emitedLight );

		const glm::vec3& emissionPower = LoadAddStore( emissionLight, indicies.first, indicies.second, receivedLight );
		LoadAddStore( reachedLight, indicies.first, indicies.second, receivedLight );
	}
}

// ============================================================================= //
// Generation
// ============================================================================= //

/**@brief G��wna funkcja generuj�ca lightmapy. Zaimplementuj w klasie pochodnej.*/
void LightmapWorkerCUDA::Generate()
{
	if( m_data->objectParts.empty() )
		return;

	checkCudaErrors( hipSetDevice( 0 ) );


	//// Chunki dla ca�ych obiek�w (nie dla podcz�ci)
	//std::vector<MemoryChunk> emissionLight;
	//std::vector<MemoryChunk> reachedLight;
	//std::vector<MemoryChunk> verticies;			// Przekszta�cone wierzcho�ki i normalne.

	thrust::device_vector<glm::vec3>		deviceEmissionLight;			// �wiat�o, kt�re dany tr�jk�t mo�e wyemitowa�.
	thrust::device_vector<glm::vec3>		deviceReachedLight;				// Kolor danego tr�jk�ta, wynikaj�cy z poch�aniania �wiat�a dochodzacego.
	thrust::device_vector<VertexFormat>		deviceVerticies;				// Wierzcho�ki meshy przekszta�cone do uk�adu wsp�rz�dnych �wiata.

	std::vector<Size> chunkOffsets;

	Prepare( deviceEmissionLight, deviceReachedLight, deviceVerticies, chunkOffsets );
	Radiosity( deviceEmissionLight, deviceReachedLight, deviceVerticies, chunkOffsets );
	BuildResult( deviceReachedLight );

	m_lightmapState = LightmapState::GenerationEnded;		// Ustaw na ko�cu funkcji, �eby w�tek g��wny m�g� zebra� wyniki.
}


class TransformVerticies : thrust::unary_function < VertexFormat, VertexNormalTexCord1 >
{
private:
	glm::mat4 modelWorldTransform;
public:
	TransformVerticies( glm::mat4 matrix )
		: modelWorldTransform( matrix ) {}

	VertexFormat operator()( VertexNormalTexCord1& vertex )
	{
		glm::vec4 position = glm::vec4( glm::make_vec3( &vertex.position ), 1.0f );
		position = modelWorldTransform * position;
		VertexFormat result;
		result.position = glm::vec3( position );
	}
};



/**@brief Przygotowuje odpowiednie struktury do generacji.*/
void LightmapWorkerCUDA::Prepare( thrust::device_vector<glm::vec3>& emissionLight, thrust::device_vector<glm::vec3>& reachedLight, thrust::device_vector<VertexFormat>& verticies, std::vector<Size>& chunkOffsets )
{
	auto& parts = m_data->objectParts;
	Size bufferLength = 0;
	std::vector<glm::vec3> materialDiffuse;

	// Budujemy tablic� offset�w w buforze dla ka�dej pocz�ci mesha.
	// Dla ka�dej cz�ci przepisujemy te� materia�.
	// Zoptymalizowa� w p�niejszych wersjach. Materia� mo�e si� powtarza� dla r�nych podmeshy.
	for( auto& part : parts )
	{
		Size chunkSize = part.verticesCount / 3;
		
		chunkOffsets.push_back( bufferLength );
		materialDiffuse.push_back( glm::vec3( part.diffuse.x, part.diffuse.y, part.diffuse.z ) );
		bufferLength += chunkSize;

		assert( materialDiffuse.size() <= MAX_MATERIALS );
	}

	// Wype�niamy bufory zerami. Dla �wiate� wektor emissionLight zostanie wype�niony p�niej odpowiednimi warto�ciami.
	emissionLight.resize( bufferLength, glm::vec3( 0.0, 0.0, 0.0 ) );
	reachedLight.resize( bufferLength, glm::vec3( 0.0, 0.0, 0.0 ) );
	verticies.resize( bufferLength * 3 );
	
	// P�tla po wszystkich podmeshach.
	// @todo zoptymalizowa� to w przysz�ej wersji. Mo�e da si� to zrobi� jako� m�drzej za jednym razem.
	for( unsigned int i = 0; i < parts.size(); ++i )
	{
		// @note Macierze DirectXMath s� row-major a glm column-major. Trzeba trnasponowa�.
		glm::mat4 transformMatrix = glm::make_mat4( (void*)&parts[i].transform );
		glm::transpose( transformMatrix );

		thrust::device_vector<VertexNormalTexCord1> sourceVerticies = GetVerticiesFromGraphicAPI( m_data->buffers[ m_data->objectParts[ i ].chunkIdx ] );
		
		// Przekszta�ca wierzcho�ki do uk�adu wsp�rz�dnych �wiata.
		thrust::transform(	sourceVerticies.begin() + m_data->objectParts[ i ].bufferOffset,
							sourceVerticies.begin() + m_data->objectParts[ i ].bufferOffset + m_data->objectParts[ i ].verticesCount,
							verticies.begin() + chunkOffsets[ i ] * 3,
							TransformVerticies( transformMatrix ) );


		// Sprawdzamy czy dany podobiekt mesha jest �wiat�em - tzn. zawiera niezerowe warto�ci emisji materia�u.
		if( parts[ i ].emissive.x != 0.0f || parts[ i ].emissive.y != 0.0f || parts[ i ].emissive.z != 0.0f )
		{
			glm::vec3 materialEmissive( parts[i].emissive.x, parts[i].emissive.y, parts[i].emissive.z );
			// Wype�niamy wektor emisji dla danego kawa�ka mesha.
			thrust::fill( emissionLight.begin() + chunkOffsets[ i ], emissionLight.begin() + chunkOffsets[ i + 1 ], materialEmissive );
		}


		UnmapVerticiesFromGraphicAPI();
	}

	// Wype�niamy pami�� sta��
	hipMemcpyToSymbol(HIP_SYMBOL( materials), materialDiffuse.data(), materialDiffuse.size() );
}

/**@brief Wykonuje algorytm radiosity.*/
void LightmapWorkerCUDA::Radiosity( thrust::device_vector<glm::vec3>& emissionLight,
									thrust::device_vector<glm::vec3>& reachedLight,
									thrust::device_vector<VertexFormat>& verticies,
									std::vector<Size>& chunkOffsets )
{
	unsigned emissionLightSize = emissionLight.size();
	unsigned verticiesSize = verticies.size();
	unsigned reachedLightSize = reachedLight.size();

	// Bufor g��boko�ci i bufor s�u��cy do indeksowania tr�jk�t�w, kt�re znajduj� si� w buforze.
	depthBufferSize = m_depthResolution * m_depthResolution;

	float*				deviceDepthBuffer;//( depthSize * sizeof( float ) );
	BufferIndexing*		deviceIndexBuffer;//( depthSize * sizeof( BufferIndexing ) );

	checkCudaErrors( hipMalloc( (void**)&deviceDepthBuffer, depthBufferSize * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**)&deviceIndexBuffer, depthBufferSize * sizeof( BufferIndexing ) ) );

	// Identyfikuje wielok�t z najwi�ksz� energi� do wyemitowania.
	std::tuple<unsigned int, unsigned int, float> emissionMax = FindMaxEmission( emissionLight, emissionLight.size() );

	// Ko�czymy generowanie, gdy najwi�ksza zgromadzona w wielok�cie energia spadnie poni�ej pewnego poziomu.
	int iterations = 0;
	while( iterations < 30 )
	{
		iterations++;

		// Czy�ci bufory 
		KernelClearDepthIndex << < ( depthBufferSize + 255 ) / 256, 256 >> > ( deviceDepthBuffer, deviceIndexBuffer );
		checkCudaErrors( hipGetLastError() );

		// Wykonuje przebieg budowania bufora Z
		DepthPass( emissionMax, verticies, deviceDepthBuffer, deviceIndexBuffer, chunkOffsets );

		// Wykonuje transfer energii pomi�dzy p�atami
		TransferPass( emissionMax, verticies, emissionLight, reachedLight, deviceIndexBuffer, chunkOffsets );

		emissionMax = FindMaxEmission( emissionLight, emissionLight.size() );
	}

	hipFree( deviceDepthBuffer );
	hipFree( deviceIndexBuffer );
}

/**W tej funkcji trzeba zebra� z wektora reachedLight dane o kolorze i przepisa�
do bufor�w razem ze wsp�rz�dnymi uv.
Dodatkowo tu b�dzie si� odbywa�o interpolowanie warto�ci w celu wyg�adzenia obrazu.
*/
void LightmapWorkerCUDA::BuildResult( thrust::device_vector<glm::vec3>& reachedLight )
{
	CoordColor*		outputBuffer;	// Pary (wsp�rz�dna uv, kolor).

	// 

	//auto& parts = m_data->objectParts;
	//auto object = parts[ 0 ].object;
	//unsigned int verticiesCount = 0;
	//unsigned int firstPartIndex = 0;

	//for( unsigned int i = 0; i < parts.size(); ++i )
	//{
	//	verticiesCount += parts[ i ].verticesCount;

	//	// Sumujemy liczb� wierzcho�k�w w obiekcie. Dopiero jak zacznie si� nowy obiekt to tworzymy bufor.
	//	unsigned int nextIdx = i + 1;
	//	if( nextIdx == parts.size() || parts[ nextIdx ].object != object )	// Pierwszy warunek wymusza, �e we�miemy tak�e ostatni obiekt
	//	{
	//		if( verticiesCount != 0 )
	//		{
	//			MemoryChunk colorMap( verticiesCount * sizeof( CoordColor ) );

	//			unsigned int verticiesOffset = 0;
	//			for( unsigned int j = firstPartIndex; j < nextIdx; ++j )
	//			{
	//				for( unsigned int k = 0; k < reachedLight[ j ].Count<glm::vec3>(); ++k )
	//				{
	//					CoordColor& colorVertex1 = colorMap.Get<CoordColor>( verticiesOffset + mul3( k ) );		// Jest 3 razy wi�cej wierzcho�k�w ni� kolor�w.
	//					glm::vec3& lightColor = reachedLight[ j ].Get<glm::vec3>( k ); /*glm::vec3( 0.5, 0.3, 0.7);*/
	//					colorVertex1.color = lightColor;
	//					colorVertex1.texCoords = m_data->verticies[ parts[ j ].chunkIdx ].Get<VertexNormalTexCord1>( parts[ j ].bufferOffset + mul3( k ) ).tex_cords;

	//					CoordColor& colorVertex2 = colorMap.Get<CoordColor>( verticiesOffset + mul3( k ) + 1 );
	//					colorVertex2.color = lightColor;
	//					colorVertex2.texCoords = m_data->verticies[ parts[ j ].chunkIdx ].Get<VertexNormalTexCord1>( parts[ j ].bufferOffset + mul3( k ) + 1 ).tex_cords;

	//					CoordColor& colorVertex3 = colorMap.Get<CoordColor>( verticiesOffset + mul3( k ) + 2 );
	//					colorVertex3.color = lightColor;
	//					colorVertex3.texCoords = m_data->verticies[ parts[ j ].chunkIdx ].Get<VertexNormalTexCord1>( parts[ j ].bufferOffset + mul3( k ) + 2 ).tex_cords;
	//				}

	//				verticiesOffset += parts[ j ].verticesCount;
	//			}

	//			m_resultData.push_back( std::move( colorMap ) );
	//		}

	//		if( nextIdx == parts.size() )		break;		// Nie ma wi�cej obiekt�w.
	//		// Zako�czyli�my przetwarzanie obiektu przechodzimy do nast�pnego.
	//		object = parts[ nextIdx ].object;
	//		firstPartIndex = nextIdx;
	//		verticiesCount = 0;
	//	}

	//}
}


/**@brief Wykonuje przebieg z z-buforem.*/
void LightmapWorkerCUDA::DepthPass( std::tuple<unsigned int, unsigned int, float>& emissionMax,
									thrust::device_vector<VertexFormat>& verticies,
									float* deviceDepthBuffer,
									BufferIndexing* deviceIndexBuffer,
									std::vector<Size>& chunkOffsets )
{
	// Pobieramy indeksy emitera.
	unsigned int idx1 = std::get<0>( emissionMax );
	unsigned int idx2 = std::get<1>( emissionMax );

	// Wyliczamy macierz widoku, kt�ra przekszta�ca wierzcho�ki do uk�adu wsp�rz�dnych zwi�zanego z emiterem.
	// Wersja na procesorze jest tylko tymczasowa.
	Triangle4 emiterPosition( &( verticies.data() + chunkOffsets[ idx1 ] * 3 ).get()[ idx2 ] );
	glm::mat4 emiterViewMatrix = EmiterViewMatrix( emiterPosition );

	// P�tla po wszystkich podobiektach. Rozwi�zanie tymczasowe do p�niejszego poprawienia przy optymalizacjach.
	for( unsigned int i = 0; chunkOffsets.size() - 1; ++i )
	{
		KernelDepthPass << < ( chunkOffsets[ i + 1 ] - chunkOffsets[ i ] + 255 ) / 256, 256 >> > ( verticies.data().get() + chunkOffsets[ i ], chunkOffsets[ i + 1 ] - chunkOffsets[ i ], i, deviceDepthBuffer, deviceIndexBuffer, emiterPosition, emiterViewMatrix );
		checkCudaErrors( hipGetLastError() );
	}
}

void LightmapWorkerCUDA::TransferPass( std::tuple<unsigned int, unsigned int, float>& emissionMax,
									   thrust::device_vector<VertexFormat>& verticies,
									   thrust::device_vector<glm::vec3>& emissionLight,
									   thrust::device_vector<glm::vec3>& reachedLight,
									   BufferIndexing* indexBuffer,
									   std::vector<Size>& chunkOffsets )
{
	// Pobieramy indeks emitera i zerujemy maksymaln� emisj�. B�dziemy szuka� nowej
	// maksymalnej warto�ci w tym przebiegu.
	std::get<2>( emissionMax ) = 0.0f;
	unsigned int idx1 = std::get<0>( emissionMax );
	unsigned int idx2 = std::get<1>( emissionMax );

	// Wyliczamy ile energii powinien dosta� ka�dy element, kt�ry znajdziemy w buforze indeks�w.
	// Zoptymalizowa�: wszystko powinno si� odbywa� na karcie bez powrot�w.
	glm::vec3 depthElementWeight = glm::vec3( static_cast<float>( 50.0 * 4.0 * XM_1DIV2PI / depthBufferSize ) );
	glm::vec3 emitedLight = ( emissionLight.data().get() + chunkOffsets[ idx1 ] )[ idx2 ];
	emitedLight = emitedLight * depthElementWeight;

	// Zerujemy �wiat�o, kt�re emiter b�dzie m�g� wyemitowa� w kolejnych iteracjach.
	( emissionLight.data().get() + chunkOffsets[ idx1 ] )[ idx2 ] = glm::vec3( 0.0, 0.0, 0.0 );

	KernelTransferPass << < ( depthBufferSize + 255 ) / 256, 256 >> > ( indexBuffer, reachedLight.data().get(), emissionLight.data().get(), emitedLight );

	checkCudaErrors( hipGetLastError() );
}



