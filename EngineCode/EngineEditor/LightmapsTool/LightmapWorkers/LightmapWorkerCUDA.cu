#include "hip/hip_runtime.h"
#include "EngineEditor/LightmapsTool/stdafx.h"
#include "EngineEditor/LightmapsTool/LightmapWorkers/LightmapWorkerCUDA.h"
#include "EngineCore/ModelsManager/DefaultAssets.h"

#include <cstring>
#include <limits>
// Trzeba oddefiniowa�, je�eli chce si� korzysta� z numeric_limits
#undef max

#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>

#include "Common/memory_leaks.h"

using namespace DirectX;

LightmapWorkerCUDA::LightmapWorkerCUDA(SceneData* sceneData)
: LightmapWorker(sceneData)
{
	m_threshold = 0.04f;
}

// ============================================================================= //
// Funkcje pomocnicze - og�lne
// ============================================================================= //

/**@brief Szybsze (miejmy nadziej�) mno�enie przez 3.*/
inline unsigned int mul3(unsigned int number)
{
	return (number << 1) + number;
}

inline float fmax(float a, float b, float c)
{
	return fmax(a, fmax(b, c));
}

/**@brief Maksymalna warto�� trzech liczb.*/
int max3(float x1, float x2, float x3)
{
	if (x1 > x2)
		return x1 > x3 ? (int)x1 : (int)x3;
	else
		return x2 > x3 ? (int)x2 : (int)x3;
}

/**@brief Minimalna warto�� trzech liczb.*/
int min3(float x1, float x2, float x3)
{
	if (x1 < x2)
		return x1 < x3 ? (int)x1 : (int)x3;
	else
		return x2 < x3 ? (int)x2 : (int)x3;
}


// ============================================================================= //
// Funkcje pomocnicze - graficzne
// ============================================================================= //

__host__ std::tuple<unsigned int, unsigned int, float> FindMaxEmission(BufferObject *emissionLight, unsigned emissionLightSize)
{
	typedef std::tuple<unsigned int, unsigned int, XMFLOAT3> Emission;

	class CompareEmissionFunction : thrust::binary_function<Emission, Emission, Emission>
	{
	public:
		Emission operator()(Emission &left, Emission &right)
		{
			XMFLOAT3 leftEmission = std::get<2>(left);
			XMFLOAT3 rightEmission = std::get<2>(right);

			float maxL = fmax(leftEmission.x, leftEmission.y, leftEmission.z);
			float maxR = fmax(rightEmission.x, rightEmission.y, rightEmission.z);

			return (maxL > maxR) ? left : right;
		}
	};

	Emission initialValue = std::make_tuple(0, 0, XMFLOAT3(-1.0f, -1.0f, -1.0f));
	Emission foundValue = thrust::reduce(emissionLight, emissionLight + emissionLightSize, initialValue, CompareEmissionFunction());

	XMFLOAT3 emissionValue = std::get<2>(foundValue);
	return std::make_tuple(std::get<0>(foundValue), std::get<1>(foundValue), fmax(emissionValue.x, emissionValue.y, emissionValue.z));
}

/**@brief Wylicza wsp�rz�dn� barycentryczn� punktu wzgl�dem kraw�dzie.
Wsp�rz�dne nie s� wyskalowane do jedynki. Nale�y je wyskalowa� kiedy si� wyliczy wszystkie.*/
__device__ static float BarycentricCoords(DirectX::XMFLOAT2& vertex1, DirectX::XMFLOAT2& vertex2, DirectX::XMINT2& point)
{
	return (vertex2.x - vertex1.x) * (point.y - vertex1.y) - (vertex2.y - vertex1.y) * (point.x - vertex1.x);
}

/*
@param[inout] receiver Tr�jk�t do zrzutowania.W tej zmiennej zwracany jest tr�jk�t wynikowy.
@return Zwraca wektor z g��boko�ciami dla ka�dego tr�jk�ta.Je�eli jaki� wierzcho�ek
znajduje si� za p�aszczyzn� rzutowania, funkcja zwraca(-1.0, -1.0, -1.0).W normalnej sytuacji
wszystkie wsp�rz�dne s� dodatnie. */
__device__ static DirectX::XMFLOAT3 HemisphereCast(Triangle4& emiter, Triangle4& receiver, XMMATRIX& emiterViewMatrix)
{
	// Poniewa� to s� wierzcho�ki, to ustawiamy komponent w na 1.
	receiver.vertex1 = XMVectorSetW(receiver.vertex1, 1.0f);
	receiver.vertex2 = XMVectorSetW(receiver.vertex2, 1.0f);
	receiver.vertex3 = XMVectorSetW(receiver.vertex3, 1.0f);

	// Transformujemy wierzcho�ki do uk�adu emitera. Dzi�ki temu xy jest pozycj� na p�aszczy�nie (na razie jeszcze nie jednostkowej)
	// a z jest odleg�o�ci� od emitera.
	XMVECTOR centerToVertex1 = XMVector4Transform(receiver.vertex1, emiterViewMatrix);
	XMVECTOR centerToVertex2 = XMVector4Transform(receiver.vertex2, emiterViewMatrix);
	XMVECTOR centerToVertex3 = XMVector4Transform(receiver.vertex3, emiterViewMatrix);

	// Wyliczenie d�ugo�ci od �rodka do wierzcho�k�w.
	XMVECTOR depth1 = XMVector3Length(centerToVertex1);
	XMVECTOR depth2 = XMVector3Length(centerToVertex2);
	XMVECTOR depth3 = XMVector3Length(centerToVertex3);

	// Normalizacja wektor�w
	receiver.vertex1 = XMVectorDivide(centerToVertex1, depth1);
	receiver.vertex2 = XMVectorDivide(centerToVertex2, depth2);
	receiver.vertex3 = XMVectorDivide(centerToVertex3, depth3);

	XMFLOAT3 depthVector;

	depthVector.x = -XMVectorGetZ(centerToVertex1);
	depthVector.y = -XMVectorGetZ(centerToVertex2);
	depthVector.z = -XMVectorGetZ(centerToVertex3);

	return depthVector;
}

/**@brief Dodaje pod podanymi indeksami w chunku element addValue.*/
__device__ inline XMFLOAT3& LoadAddStore(BufferObject *chunk, unsigned int i, unsigned int j, XMVECTOR addValue)
{
	auto& lightValueRef = chunk[i].Get<XMFLOAT3>(j);
	XMVECTOR lightValue = XMLoadFloat3(&lightValueRef);
	lightValue = XMVectorAdd(addValue, lightValue);
	XMStoreFloat3(&lightValueRef, lightValue);

	return lightValueRef;
}

/**@brief Rasteryzuje podany tr�jk�t. Sprawdza czy jest bli�ej emitera od ostatnio
wpisanego tr�jk�ta na podstawie bufora g��boko�ci i je�eli tak, to zapisuje nowy tr�jk�t.

Rasteryzacja zrobiona mniej wi�cej na podstawie strony:
https://fgiesen.wordpress.com/2013/02/06/the-barycentric-conspirac/
Warto przeczyta� te� kilka temat�w pokrewnych. Na razie implementuj� wersj� niewydajn�.
Mo�e potem przerobi� na lepsz�. W ka�dym razie warto zauwa�y�, �e ten algorytm jest bardzo dobry
do zr�wnoleglania, mo�e da si� to jako� sprytnie wykorzysta�.

@param[in] triangle Tr�jk�t do rasteryzacji.
@param[in] depths Odleg�o�ci (g��boko�ci) kolejnych wierzcho�k�w tr�jk�ta od emitera. (Tu jest ma�e oszustwo,
zobaczymy jak b�dzie si� sprawdza�, odleg�o�ci s� liczone do �rodka emitera).
@param[in] chunkIdx Indeks MemoryChunka do wpisania do bufora indeks�w.
@param[in] triangleIdx Indeks tr�jk�ta, kt�ry trafi do bufora indeks�w.
@param[in] depthBuffer Bufor g��boko�ci.
@param[in] indexBuffer Bufor indeks�w.*/
__device__ static void RasterizeTriangle(const Triangle4& triangle,
	DirectX::XMFLOAT3* depths,
	unsigned int chunkIdx,
	unsigned int triangleIdx,
	BufferObject *depthBuffer,
	BufferIndexing *indexBuffer,
	unsigned int m_depthResolution)
{
	XMFLOAT2A triangles[3];
	XMStoreFloat2A(&triangles[0], triangle.vertex1);
	XMStoreFloat2A(&triangles[1], triangle.vertex2);
	XMStoreFloat2A(&triangles[2], triangle.vertex3);

	// Wyliczamy prostok�t ograniczaj�cy.
	int minX = min3(triangles[0].x, triangles[1].x, triangles[2].x);
	int minY = min3(triangles[0].y, triangles[1].y, triangles[2].y);
	int maxX = max3(triangles[0].x, triangles[1].x, triangles[2].x);
	int maxY = max3(triangles[0].y, triangles[1].y, triangles[2].y);

	// Iterujemy po wszystkich elementach prostok�ta ograniczaj�cego.
	// Sprawdzamy czy dany punkt nale�y do tr�jk�ta. Je�eli nale�y
	// zagl�damy do z bufora.
	XMINT2 point;
	for (point.y = minY; point.y <= maxY; point.y++)
	{
		for (point.x = minX; point.x <= maxX; point.x++)
		{
			float w0 = BarycentricCoords(triangles[1], triangles[2], point);
			float w1 = BarycentricCoords(triangles[2], triangles[0], point);
			float w2 = BarycentricCoords(triangles[0], triangles[1], point);

			if (w0 >= 0 && w1 >= 0 && w2 >= 0)	// Warunek na to, �e punkt jest wewn�trz tr�jk�ta.
			{
				float sum = w0 + w1 + w2;
				w0 = w0 / sum;
				w1 = w1 / sum;
				w2 = w2 / sum;

				int index = point.y * m_depthResolution + point.x;
				//float pointInverseDepth = w0 / ((float*)depths)[ 0 ] + w1 / ((float*)depths)[ 1 ] + w2 / ((float*)depths)[ 2 ];
				//float pointDepth = 1.0f / pointInverseDepth;
				float pointDepth = ((float*)depths)[0] * w0 + ((float*)depths)[1] * w1 + ((float*)depths)[2] * w2;
				float& bufferDepth = depthBuffer.Get<float>(index);
				if (bufferDepth > pointDepth && pointDepth > 0.0f)
				{
					BufferIndexing& indicies = indexBuffer.Get<BufferIndexing>(index);
					indicies.first = chunkIdx;
					indicies.second = triangleIdx;
					bufferDepth = pointDepth;
				}
			}
		}
	}

}

/**@brief Transformuje wierzcho�ki w przedziale [-1,1] do wsp�rzednych bufora g��boko�ci.*/
__device__ static void HemisphereViewport(Triangle4& receiver, unsigned int m_depthResolution)
{
	XMVECTOR depthResolution = XMVectorReplicate((float)m_depthResolution / 2.0f);
	receiver.vertex1 = XMVectorMultiplyAdd(receiver.vertex1, depthResolution, depthResolution);
	receiver.vertex2 = XMVectorMultiplyAdd(receiver.vertex2, depthResolution, depthResolution);
	receiver.vertex3 = XMVectorMultiplyAdd(receiver.vertex3, depthResolution, depthResolution);
}

/**@brief Tworzy macierz przekszta�caj�c� wierzcho�ki do uk�adu odniesienia emitera.*/
__host__ DirectX::XMMATRIX LightmapWorkerCUDA::EmiterViewMatrix(Triangle4& emiter)
{
	// �rodek ci�ko�ci tr�jk�ta przyjujemy za �rodek emitera.
	XMVECTOR emiterCenter = XMVectorAdd(emiter.vertex1, emiter.vertex2);
	emiterCenter = XMVectorAdd(emiterCenter, emiter.vertex3);
	emiterCenter = XMVectorDivide(emiterCenter, XMVectorReplicate(3.0f));

	// Wektor prostopad�y do p�aszczyzny, na kt�rej le�� tr�jk�ty.
	XMVECTOR edge12 = XMVectorSubtract(emiter.vertex2, emiter.vertex1);
	XMVECTOR edge13 = XMVectorSubtract(emiter.vertex3, emiter.vertex1);
	XMVECTOR normal = XMVector3Normalize(XMVector3Cross(edge12, edge13));

	// Ustalamy dowolnie (byle ortogonalnie) kierunek "w g�r�".
	//XMVECTOR upDirection = XMVector3Normalize( XMVector3Cross( edge12, normal ) );
	XMVECTOR upDirection = XMVector3Orthogonal(normal);

	// Tworzymy macierz widoku.
	normal = XMVectorSetW(normal, 0.0f);
	emiterCenter = XMVectorSetW(emiterCenter, 1.0f);
	upDirection = XMVectorSetW(upDirection, 0.0f);

	XMMATRIX viewMatrix = XMMatrixLookToRH(emiterCenter, normal, upDirection);
	return viewMatrix;
}

// ============================================================================= //
// Generation - kernele
// ============================================================================= //

/**@brief Czy�ci bufory g��boko�ci i indeks�w.*/
__global__ static void kernel_ClearDepthIndex(float *depthBuffer, BufferIndexing *indexBuffer)
{
	int i = blockIdx * blockDim.x + threadIdx.x;

	depthBuffer[i] = std::numeric_limits<float>::max();
	indexBuffer[i].first = INVALID_INDEX;
	indexBuffer[i].second = INVALID_INDEX;
}

__global__ static void kernel_DepthPass(BufferObject *verticies, unsigned verticiesSize,
	BufferObject *depthBuffer, BufferIndexing *indexBuffer, unsigned int idx1, unsigned int idx2, Triangle4 emiterPosition, XMMATRIX emiterViewMatrix)
{
	// Pierwsza wsp�rz�dna wierzcho�ka
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (unsigned int j = 0; j < verticies[i].Count<VertexFormat>() / 3; ++j)
	{
		if (idx2 == j && idx1 == i)
			return;	// Nie pr�bujmy rzutowa� emitera na samego siebie.

		// Wyliczamy pozycj� odbiorcy w uk�adzie wsp�rz�dnych bufora g��boko�ci.
		Triangle4 receiverPosition(&verticies[i].Get<VertexFormat>(mul3(j)));
		XMFLOAT3 receiverDepths = HemisphereCast(emiterPosition, receiverPosition, emiterViewMatrix);

		if (receiverDepths.x < 0.0f || receiverDepths.y < 0.0f || receiverDepths.z < 0.0f)
			return;  // Tr�jk�t jest cz�ciowo za p�aszczyzn�. Musimy go odrzuci�.

		HemisphereViewport(receiverPosition);
		RasterizeTriangle(receiverPosition, &receiverDepths, i, j, depthBuffer, indexBuffer);
	}
}

__global__ static void kernel_TransferPass(SceneObjectData *objectParts, BufferIndexing *indexBuffer,
	BufferObject *reachedLight, BufferObject *emissionLight, XMVECTOR emitedLight)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	BufferIndexing indicies = indexBuffer[i];
	// Tylko piksele na kole powinny dosta� dawk� energii, a bufor jest kwadratem.
	// Dlatego elementy, na kt�re nie zosta� zrzutowany �aden tr�jk�t zosta�y zainicjowane warto�ci� INVALID_INDEX.
	if (indicies.first != INVALID_INDEX)
	{
		XMVECTOR materialDiffuse = XMLoadFloat4(&objectParts[indicies.first].diffuse);
		XMVECTOR receivedLight = XMVectorMultiply(materialDiffuse, emitedLight);

		const XMFLOAT3& emissionPower = LoadAddStore(emissionLight, indicies.first, indicies.second, receivedLight);
		LoadAddStore(reachedLight, indicies.first, indicies.second, receivedLight);
	}
}

// ============================================================================= //
// Generation
// ============================================================================= //

/**@brief G��wna funkcja generuj�ca lightmapy. Zaimplementuj w klasie pochodnej.*/
void LightmapWorkerCUDA::Generate()
{
	if (m_data->objectParts.empty())
		return;

	checkCudaErrors(hipSetDevice(0));


	// Chunki dla ca�ych obiek�w (nie dla podcz�ci)
	std::vector<MemoryChunk> emissionLight;
	std::vector<MemoryChunk> reachedLight;
	std::vector<MemoryChunk> verticies;			// Przekszta�cone wierzcho�ki i normalne.

	Prepare(emissionLight, reachedLight, verticies);
	Radiosity(emissionLight, reachedLight, verticies);
	BuildResult(reachedLight);

	m_lightmapState = LightmapState::GenerationEnded;		// Ustaw na ko�cu funkcji, �eby w�tek g��wny m�g� zebra� wyniki.
}

/**@brief Przygotowuje odpowiednie struktury do generacji.*/
void LightmapWorkerCUDA::Prepare(std::vector<MemoryChunk>& emissionLight, std::vector<MemoryChunk>& reachedLight, std::vector<MemoryChunk>& verticies)
{
	auto& parts = m_data->objectParts;
	for (auto& part : parts)
	{
		unsigned int chunkSize = part.verticesCount * sizeof(XMFLOAT3) / 3;

		MemoryChunk emissionChunk(chunkSize);
		MemoryChunk reachedChunk(chunkSize);
		MemoryChunk verticiesChunk(part.verticesCount * sizeof(VertexFormat));

		memset(reachedChunk.GetMemory<void>(), 0, chunkSize);
		memset(emissionChunk.GetMemory<void>(), 0, chunkSize);		// Potem wype�nimy warto�ciami emisji w p�tli.

		XMMATRIX transformMatrix = XMLoadFloat4x4(&part.transform);

		// Transform positions
		XMVector3TransformCoordStream(&verticiesChunk.Get<VertexFormat>(0).position,
			sizeof(VertexFormat),
			&m_data->verticies[part.chunkIdx].Get<VertexNormalTexCord1>(part.bufferOffset).position,
			sizeof(VertexNormalTexCord1),
			part.verticesCount,
			transformMatrix);

		// Transform normals
		XMVector3TransformNormalStream(&verticiesChunk.Get<VertexFormat>(0).normal,
			sizeof(VertexFormat),
			&m_data->verticies[part.chunkIdx].Get<VertexNormalTexCord1>(part.bufferOffset).normal,
			sizeof(VertexNormalTexCord1),
			part.verticesCount,
			transformMatrix);

		if (part.emissive.x != 0.0f || part.emissive.y != 0.0f || part.emissive.z != 0.0f)
		{
			XMFLOAT3 materialEmissive(part.emissive.x, part.emissive.y, part.emissive.z);

			unsigned int trianglesCount = part.verticesCount / 3;

			// To takie skalowanie na oko troszk�. W ka�dym razie zak�adamy, �e �wiat�o o mocy (1,1,1) powinno wypromieniowa�
			// tyle samo energii niezale�nie od ilo�ci wierzcho�k�w. W przysz�o�ci trzeba b�dzie jako� pewnie uwzgl�dnia� powierzchni�
			// tr�jk�t�w zar�wno emituj�cych jak i odbieracj�cych. Tych oblicze� widz� robi si� coraz wi�cej.
			//float scaleFactor = 1.0f/*(float)m_depthResolution *// (float)trianglesCount;
			//materialEmissive.x = materialEmissive.x * scaleFactor;
			//materialEmissive.y = materialEmissive.y * scaleFactor;
			//materialEmissive.z = materialEmissive.z * scaleFactor;

			for (unsigned int k = 0; k < trianglesCount; ++k)
				emissionChunk.Get<XMFLOAT3>(k) = materialEmissive;
		}

		emissionLight.push_back(std::move(emissionChunk));
		reachedLight.push_back(std::move(reachedChunk));
		verticies.push_back(std::move(verticiesChunk));
	}
}

/**@brief Wykonuje algorytm radiosity.*/
void LightmapWorkerCUDA::Radiosity(std::vector<MemoryChunk>& emissionLight,
	std::vector<MemoryChunk>& reachedLight,
	std::vector<MemoryChunk>& verticies)
{
	unsigned emissionLightSize = emissionLight.size();
	unsigned verticiesSize = verticies.size();
	unsigned reachedLightSize = reachedLight.size();

	// Bufor g��boko�ci i bufor s�u��cy do indeksowania tr�jk�t�w, kt�re znajduj� si� w buforze.
	unsigned int		depthSize = m_depthResolution * m_depthResolution;
	MemoryChunk			depthBuffer(depthSize * sizeof(float));
	MemoryChunk			indexBuffer(depthSize * sizeof(BufferIndexing));
		
	// Kolekcje znajduj�ce si� na GPU nie wymagaj� kopiowania,
	//  tylko np. hipGraphicsMapResources
	BufferObject *dev_verticies;
	BufferObject *dev_depthBuffer;
	BufferIndexing *dev_indexBuffer;
	BufferObject *dev_reachedLight;
	BufferObject *dev_emissionLight;
	// FIXME: uzupe�ni� definicje tablic dev_

	// Identyfikuje wielok�t z najwi�ksz� energi� do wyemitowania.
	std::tuple<unsigned int, unsigned int, float> emissionMax = FindMaxEmission(dev_emissionLight, emissionLightSize);

	// Ko�czymy generowanie, gdy najwi�ksza zgromadzona w wielok�cie energia spadnie poni�ej pewnego poziomu.
	int iterations = 0;

	checkCudaErrors(hipSetDevice(0));

	while (iterations < 30)
	{
		iterations++;

		// Czy�ci bufory 
		//  FixMe: transformacja BufferObject -> float
		kernel_ClearDepthIndex << < (depthSize + 255) / 256, 256 >> > (dev_depthBuffer, dev_indexBuffer);
		checkCudaErrors(hipGetLastError());

		// Wykonuje przebieg budowania bufora Z
		DepthPass(emissionMax, dev_verticies, verticiesSize, dev_depthBuffer, dev_indexBuffer);

		// Wykonuje transfer energii pomi�dzy p�atami
		TransferPass(emissionMax, dev_verticies, verticiesSize, dev_emissionLight, dev_reachedLight, dev_indexBuffer);

		emissionMax = FindMaxEmission(dev_emissionLight, emissionLightSize);
	}
}

/**@brief Tworzy tablic� color - uv, kt�ra zostanie potem u�yta do wyrenderowania lightmapy.
Wszystkie chunki podzielone wcze�niej na podcz�ci s� teraz ��czone. Ka�dy obiekt b�dzie mia� sw�j bufor.*/
void LightmapWorkerCUDA::BuildResult(std::vector<MemoryChunk>& reachedLight)
{
	auto& parts = m_data->objectParts;
	auto object = parts[0].object;
	unsigned int verticiesCount = 0;
	unsigned int firstPartIndex = 0;

	for (unsigned int i = 0; i < parts.size(); ++i)
	{
		verticiesCount += parts[i].verticesCount;

		// Sumujemy liczb� wierzcho�k�w w obiekcie. Dopiero jak zacznie si� nowy obiekt to tworzymy bufor.
		unsigned int nextIdx = i + 1;
		if (nextIdx == parts.size() || parts[nextIdx].object != object)	// Pierwszy warunek wymusza, �e we�miemy tak�e ostatni obiekt
		{
			if (verticiesCount != 0)
			{
				MemoryChunk colorMap(verticiesCount * sizeof(CoordColor));

				unsigned int verticiesOffset = 0;
				for (unsigned int j = firstPartIndex; j < nextIdx; ++j)
				{
					for (unsigned int k = 0; k < reachedLight[j].Count<XMFLOAT3>(); ++k)
					{
						CoordColor& colorVertex1 = colorMap.Get<CoordColor>(verticiesOffset + mul3(k));		// Jest 3 razy wi�cej wierzcho�k�w ni� kolor�w.
						XMFLOAT3& lightColor = reachedLight[j].Get<XMFLOAT3>(k); /*XMFLOAT3( 0.5, 0.3, 0.7);*/
						colorVertex1.color = lightColor;
						colorVertex1.texCoords = m_data->verticies[parts[j].chunkIdx].Get<VertexNormalTexCord1>(parts[j].bufferOffset + mul3(k)).tex_cords;

						CoordColor& colorVertex2 = colorMap.Get<CoordColor>(verticiesOffset + mul3(k) + 1);
						colorVertex2.color = lightColor;
						colorVertex2.texCoords = m_data->verticies[parts[j].chunkIdx].Get<VertexNormalTexCord1>(parts[j].bufferOffset + mul3(k) + 1).tex_cords;

						CoordColor& colorVertex3 = colorMap.Get<CoordColor>(verticiesOffset + mul3(k) + 2);
						colorVertex3.color = lightColor;
						colorVertex3.texCoords = m_data->verticies[parts[j].chunkIdx].Get<VertexNormalTexCord1>(parts[j].bufferOffset + mul3(k) + 2).tex_cords;
					}

					verticiesOffset += parts[j].verticesCount;
				}

				m_resultData.push_back(std::move(colorMap));
			}

			if (nextIdx == parts.size())		break;		// Nie ma wi�cej obiekt�w.
			// Zako�czyli�my przetwarzanie obiektu przechodzimy do nast�pnego.
			object = parts[nextIdx].object;
			firstPartIndex = nextIdx;
			verticiesCount = 0;
		}

	}
}

/**@brief Wykonuje przebieg z z-buforem.*/
void LightmapWorkerCUDA::DepthPass(std::tuple<unsigned int, unsigned int, float>& emissionMax,
	BufferObject *dev_verticies, unsigned verticiesSize, BufferObject *dev_depthBuffer, BufferObject *dev_indexBuffer)
{
	// Pobieramy indeksy emitera.
	unsigned int idx1 = std::get<0>(emissionMax);
	unsigned int idx2 = std::get<1>(emissionMax);

	uint32 depthFailCounter = 0;
	uint32 triangleCounter = 0;

	// Liczmymy pozycj� emitera, u�rednion� normaln� i uk�ad wsp�rz�dnych zwi�zany z emiterem dla z-bufora.
	Triangle4 emiterPosition(&verticies[idx1].Get<VertexFormat>(mul3(idx2)));
	XMMATRIX emiterViewMatrix = EmiterViewMatrix(emiterPosition);

	kernel_DepthPass << < (verticiesSize + 255) / 256, 256 >> > (dev_depthBuffer, dev_indexBuffer, idx1, idx2, emiterPosition, emiterViewMatrix);
	checkCudaErrors(hipGetLastError());
}

void LightmapWorkerCUDA::TransferPass(std::tuple<unsigned int, unsigned int, float>& emissionMax,
	BufferObject *verticies,
	unsigned verticiesSize,
	BufferObject *emissionLight,
	BufferObject *reachedLight,
	BufferIndexing *indexBuffer)
{
	// Pobieramy indeks emitera i zerujemy maksymaln� emisj�. B�dziemy szuka� nowej
	// maksymalnej warto�ci w tym przebiegu.
	std::get<2>(emissionMax) = 0.0f;
	unsigned int idx1 = std::get<0>(emissionMax);
	unsigned int idx2 = std::get<1>(emissionMax);

	unsigned int depthSize = m_depthResolution * m_depthResolution;

	// Wyliczamy ile energii powinien dosta� ka�dy element, kt�ry znajdziemy w buforze indeks�w.
	XMVECTOR depthElementWeight = XMVectorReplicate(static_cast<float>(50.0 * 4.0 * XM_1DIV2PI / depthSize));
	XMVECTOR emitedLight = XMLoadFloat3(&emissionLight[idx1].Get<XMFLOAT3>(idx2));
	emitedLight = XMVectorMultiply(emitedLight, depthElementWeight);

	// Zerujemy �wiat�o, kt�re emiter b�dzie m�g� wyemitowa� w kolejnych iteracjach.
	XMStoreFloat3(&emissionLight[idx1].Get<XMFLOAT3>(idx2), XMVectorZero());

	kernel_TransferPass << < (depthSize + 255) / 256, 256 >> > (&m_data->objectParts, indexBuffer, 
		reachedLight, emissionLight, emitedLight);

	checkCudaErrors(hipGetLastError());
}


/**@brief Wylicza wsp�rz�dne tr�jk�ta zrzutowanego na p�aszczyzn� ko�a.
Wsp�rz�dne s� dwuwymiarowe, co pozwala �atwo przej�� z nich na indeksy do bufora
g��boko�ci.

/**@brief Wyliczamy wektory r�wnoleg�e do p�aszczyzny i wzajemnie ortogonalne.

Pos�u�� one za uk�ad wsp�rz�dnych, w kt�rym b�dzie wyra�ony rzut tr�jk�ta.
W zasadzie ich orientacja jest dowolna, wa�ne �eby by�y stosowane te same
wektory do wszystkich rzut�w.*/
Triangle4 LightmapWorkerCUDA::EmiterCoordinatesSystem(Triangle4& emiter)
{
	XMVECTOR edge12 = XMVectorSubtract(emiter.vertex2, emiter.vertex1);
	XMVECTOR edge13 = XMVectorSubtract(emiter.vertex3, emiter.vertex1);

	// Wyliczamy najpierw wektor normalny do p�aszczyzny, z potem tworzymy 2 kolejne wektory ortogonalne.
	Triangle4 CoordSystem;
	CoordSystem.vertex1 = XMVector3Normalize(XMVector3Cross(edge13, edge12));
	CoordSystem.vertex2 = XMVector3Normalize(edge12);															// Jedn� z osi wybieramy totalnie dowolnie, wa�ne by by�a na p�aszczy�nie.
	CoordSystem.vertex3 = XMVector3Normalize(XMVector3Cross(CoordSystem.vertex2, CoordSystem.vertex1));		// Obliczamy wektor prostopad�y do normalnej i pierwszej osi.

	return CoordSystem;
}

/**@brief Wylicza rzut punktu na p�aszczyzn�.

@param[in] point Punkt wyra�ony jako wektor znormalizowany od �rodka hemisfery do rzutowanego punktu.
@param[in] coordsSystem Opisuje uk�ad wsp�rz�dnych w jakim ma by� wyra�ony zwracany punkt.
@return Funkcja zwaraca dwuwymiarowy wektor oznaczaj�cy punkt we wsp�rzednych w podanym
uk�adzie wsp�rz�dnych.*/
XMVECTOR LightmapWorkerCUDA::ProjectPointToPlane(XMVECTOR point, const Triangle4& coordsSystem)
{
	XMVECTOR axisU = XMVector3Dot(coordsSystem.vertex2, point);
	XMVECTOR axisV = XMVector3Dot(coordsSystem.vertex3, point);
	XMVECTOR result = XMVectorMergeXY(axisU, axisV);

	return result;
}



